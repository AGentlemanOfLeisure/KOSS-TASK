#include <iostream>
#include <hip/hip_runtime.h>

// Matrix size
#define N 32
#define BLOCK_SIZE 16

// CUDA Kernel for matrix addition
__global__ void addMatrices(int *A, int *B, int *C, int width) {
    // Calculate row and column index of the element in the result matrix
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < width && col < width) {
        int index = row * width + col;
        C[index] = A[index] + B[index];
    }
}

int main() {
    // Matrix size
    int width = N;
    int size = width * width * sizeof(int);

    // Allocate memory for matrices A, B, and C
    int *A, *B, *C;
    int *d_A, *d_B, *d_C;
    
    // Allocate memory on the host
    A = (int*)malloc(size);
    B = (int*)malloc(size);
    C = (int*)malloc(size);

    // Initialize matrices A and B
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            A[i * width + j] = i;  // A[i][j] = i
            B[i * width + j] = j;  // B[i][j] = j
        }
    }

    // Allocate memory on the device
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy matrices A and B from host to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (width + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Create CUDA events to measure time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start time
    hipEventRecord(start, 0);

    // Launch the kernel
    addMatrices<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, width);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }

    // Record the stop time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy the result matrix C from device to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Print the result matrix C
    std::cout << "Matrix C (A + B):" << std::endl;
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            std::cout << C[i * width + j] << " ";
        }
        std::cout << std::endl;
    }

    // Print the time taken to execute the kernel
    std::cout << "Time taken for kernel execution: " << milliseconds << " ms" << std::endl;

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(A);
    free(B);
    free(C);

    return 0;
}
